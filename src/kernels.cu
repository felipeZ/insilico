
#include <hip/hip_runtime.h>

// Sum two vectors
__global__ void sumArraysOnGPU(float *A, float *B, float *C) {
  int i = threadIdx.x;
  C[i] = A[i] + B[i];
}

void callSumOnGPU(float *dA, float *dB, float *dC) {
  sumArraysOnGPU<<< 1, 1 >>>(dA, dB, dC);  
}
