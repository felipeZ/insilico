
#include <hip/hip_runtime.h>

// Sum two vectors
template <typename T>
__global__ void sumArraysOnGPU(T *A, T *B, T *C) {
  int i = threadIdx.x;
  C[i] = A[i] + B[i];
}

template <typename T>
void callSumOnGPU(T *dA, T *dB, T *dC, int grids, int blocks) {
  sumArraysOnGPU<<< grids, blocks >>>(dA, dB, dC);  
}

// Instantiate the kernel template
template void callSumOnGPU<float>(float *dA, float *dB, float *dC, int m, int n);
template void callSumOnGPU<double>(double *dA, double *dB, double *dC, int m, int n);